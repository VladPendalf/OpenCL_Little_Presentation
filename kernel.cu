#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <fstream>
#include <thread>
#include <chrono>
#include <iostream>
#include <vector>

#define VECTOR_SIZE 100'000'000

std::chrono::steady_clock::time_point begin, end;
std::chrono::microseconds diff_time;

hipError_t addWithCuda(float* c, float* a, float* b, unsigned int size);

__global__ void addKernel(float *c, float *a, float *b)
{
    //int i = threadIdx.x;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    c[i] = a[i] + b[i];
}

void ReadToIn(std::string inputFilename,
              std::vector<float>* vecToData)
{
    std::ifstream in(inputFilename);

    if (!in.is_open())
        std::cout << inputFilename << " not open";
    
    auto size = vecToData->size();
    vecToData->insert(vecToData->begin(),
                     std::istream_iterator<float>(in),
                     std::istream_iterator<float>());
}

void ReadToOut(std::string outFilename,
               std::vector<float> data)
{
    std::ofstream out(outFilename, std::ofstream::out);

    auto size = data.size();
    for (size_t i = 0; i < size; ++i)
        out << data[i] << "\n";
}

int main()
{
    srand(time(0));
    std::vector<float> a(VECTOR_SIZE,0);
    std::vector<float> b(VECTOR_SIZE,0);
    std::vector<float> c(VECTOR_SIZE,0);

    std::thread thrA(ReadToIn, "Cuda_A2.txt", &a),
        thrB(ReadToIn, "Cuda_B2.txt", &b);
    thrA.join();
    thrB.join();


    //for (int i = 0; i < VECTOR_SIZE; i++)
    //{
    //    a[i] = ((rand() % 200001) - 10000); //[-100'000:100'000]
    //    b[i] = ((rand() % 200001) - 10000);
    //}

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c.data(), a.data(), b.data(), VECTOR_SIZE);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    std::thread thrC(ReadToOut, "Cuda_C2.txt", c);
    thrC.join();
    std::cout << "\nOutput end\n";

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(float* c, float* a, float* b, unsigned int size)
{
    float *dev_a = 0;
    float *dev_b = 0;
    float *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    begin = std::chrono::steady_clock::now();
    //<<<AllSizeData/(2*count_of_cores_in_your_GPU), (2*count_of_cores_in_your_GPU)>>>
    addKernel<<<VECTOR_SIZE/1000, 1000>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }
    end = std::chrono::steady_clock::now();
    diff_time = std::chrono::duration_cast<std::chrono::microseconds>(end - begin);
    std::cout << "\nTime = " << diff_time.count() << "microseconds\n";

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
